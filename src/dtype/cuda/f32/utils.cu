
#include <hip/hip_runtime.h>
#include <stdio.h>


hipError_t set_gpu(int dev) {
    int dev_count = 0;
    hipError_t err_count = hipGetDeviceCount(&dev_count);
    if (err_count != hipSuccess) {
        return err_count;
    }
    if (dev >= dev_count) {
        return cudaErrorDevicesUnavailable;
    }
    
    hipError_t err_set = hipSetDevice(dev);
    return err_set;
}